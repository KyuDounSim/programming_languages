
#include <hip/hip_runtime.h>
#include <iostream>

// Calculate the multiplication of two 32*32 matrices A and B on gpu and store the result in C.
// Each block calculate one element of C.
__global__ void Mul(int* d_A, int* d_B, int* d_C) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int num_threads = blockDim.x * gridDim.x;

    for (int i = tid; i < 32 * 32; i += num_threads) {
       	int row = i / 32;
       	int col = i % 32;
		d_C[row * 32 + col] = 0;

		// sum of d_A(row, i) * d_B(i, col)
       	for(int j = 0; j < 32; j++) {
       		d_C[row * 32 + col] += d_A[row * 32 + j] * d_B[j * 32 + col];
       	}
	}
}

int main() {
	int *A = (int*)malloc(32 * 32 * sizeof(int));
	int *B = (int*)malloc(32 * 32 * sizeof(int));

	for(int i = 0; i < 32 * 32; i++) {
        	A[i] = 1;
        	B[i] = 1;
	}

	//Allocate the memory in GPU to store the content of A,B,C
	int *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, 32 * 32 * sizeof(int));
	hipMalloc((void **)&d_B, 32 * 32 * sizeof(int));
	hipMalloc((void **)&d_C, 32 * 32 * sizeof(int));

	//Copy A, B to d_A,d_B
	hipMemcpy(d_A, A, 32 * 32 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, 32 * 32 * sizeof(int), hipMemcpyHostToDevice);

	Mul<<<4,32>>>(d_A, d_B, d_C);

	int *C = (int*)malloc(32 * 32 * sizeof(int));
	hipMemcpy(C, d_C, 32 * 32 * sizeof(int), hipMemcpyDeviceToHost);

	//print the result
	for(int i = 0; i < 32; i++) {
		for(int j = 0; j < 32; j++) {
			std::cout << C[i * 32 + j] << " ";
		}
		std::cout << std::endl;
	}

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(A);
	free(B);
	free(C);

	return 0;
}

