/*
 * Do not change this file
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cassert>
#include <cstring>
#include <string>
#include <chrono>
#include <cstdlib>
#include <ctime>
#define MAX_LENGTH 4096

/**
 * Read file, save edges to array (x_x) and 
 * record the size of each type of edge array (x_x_count).
 */
namespace utils {
	int target;
	int len;
	int *list;

	int read_file(std::string filename) {
		std::ifstream inputf(filename, std::ifstream::in);
		len = 0;
		list = (int*)malloc(sizeof(int) * MAX_LENGTH);
		if(inputf) {
			while (!inputf.eof()){
				inputf >> list[len];
				len++;
			}
		} else {
			return -1;
		}
		inputf.close();
		return 0;
	}

	int read_target() {
		std::cout << "Your Input Target n : " <<std::endl;
		std::cin >> utils::target;
		std::cout << "FrequencyPrefixSum of target " << utils::target << " array is output to out.txt." << std::endl;
		std::cout << "Please compare the out.txt with the groundtruth answer_for_n_is_2.txt by command:" << std::endl;
		std::cout << "\tdiff out.txt answer_for_n_is_2.txt" << std::endl;
		return 0;
	}

	int write_file(std::string filename, int *out){
		std::ofstream outputf(filename, std::ofstream::out);
		if(outputf.is_open()){
			for(int i = 0; i < len; i++){
				outputf << out[i] << std::endl;
			}
		} else {
			return -1;
		}
		outputf.close();
		return 0;
	}
}

/**
 * Global function: prefix sum
 * d_in:              original array
 * d_out:             prefix sum array (need to be allocated before)
 * numElems:          the sum of the array.
 */
__global__ void prefix_sum_kernel(int *d_out, int *d_in, int numElems) {
	unsigned int d_hist_idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (d_hist_idx >= numElems)
	{
		return;
	}

	unsigned int cdf_val = 0;
	for (int i = 0; i <= d_hist_idx; ++i)
	{
		cdf_val = cdf_val + d_in[i];
	}
	d_out[d_hist_idx] = cdf_val;
}

__global__ void improved_prefix_sum_kernel(int *out, int *in, int n)
{
	__shared__ int temp[2049];

	int threadId = threadIdx.x;
	int offset = 1;

	//load input into shared memory
	temp[2 * threadId] = in[2 * threadId];
	temp[2 * threadId + 1] = in[2 * threadId + 1];
	__syncthreads();

	for(int d = n/2; d > 0; d /= 2) // build sum in place up the tree
	{
		__syncthreads();
		if(threadId < d)
		{
			int ai = offset * (2 * threadId + 1) - 1;
			int bi = offset * (2 * threadId + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if(threadId == 0) // clear the last element
		temp[n-1] = 0;

	for(int d = 1; d < n; d *= 2)
	{
		offset /= 2;
		__syncthreads();

		if(threadId < d)
		{
			int ai = offset * (2 * threadId + 1) - 1;
			int bi = offset * (2 * threadId + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	out[2 * threadId] = temp[2 * threadId + 1];
	out[2 * threadId + 1] = temp[2 * threadId + 2];
	if (threadId == 0) {
		out[n - 1] = out[n - 2] + in[n - 1];
	}
}

__global__ void map_kernel(int *out, int *in, int numElems, int target_n){
	unsigned int d_hist_idx = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int num_threads = blockDim.x * gridDim.x;

	for (int i = d_hist_idx; i < numElems; i += num_threads)
	{
		if(target_n == in[i]){
			out[i] = 1;
		} else {
			out[i] = 0;
		}
	}
}

int main(int argc, char **argv) {
    assert(argc == 3 && "Input format error!");
    std::string filename = argv[1];
    std::string out_filename = argv[2];

    assert(utils::read_file(
        	filename
    		) == 0
		);
	assert(utils::read_target() == 0);

	dim3 grid(1);
	dim3 block(1024);

	int numElems = utils::len;
	int *h_in = utils::list;
	int *h_out = (int*)malloc(sizeof(int) * numElems);
	int *d_in;
	int *d_out;

	hipMalloc((void**)&d_in, sizeof(int) * numElems);
	hipMalloc((void**)&d_out, sizeof(int) * numElems);
	hipMemcpy(d_in, h_in, sizeof(int) * numElems, hipMemcpyHostToDevice);

	/*begin--- fill your code here to achieve functionality of frequencyPrefixSum */

    auto t_start = std::chrono::high_resolution_clock::now();

    hipEvent_t cuda_start, cuda_end;
    hipEventCreate(&cuda_start);
    hipEventCreate(&cuda_end);
    float naive_kernel_time;
    float improved_kernel_time;
    hipEventRecord(cuda_start);

	prefix_sum_kernel<<<grid, block>>>(
			d_out,
			d_in,
			numElems);

    hipEventRecord(cuda_end);

    hipEventSynchronize(cuda_start);
    hipEventSynchronize(cuda_end);
    hipEventElapsedTime(&naive_kernel_time, cuda_start, cuda_end);
	hipDeviceSynchronize();
	hipMemcpy(h_out, d_out, sizeof(int) * numElems, hipMemcpyDeviceToHost);
	assert(
			utils::write_file(
					"naive_out.txt",
					h_out
			) == 0
	);

	/*end ---- fill your code here to achieve functionality of frequencyPrefixSum */

	hipMemcpy(h_out, d_out, sizeof(int) * numElems, hipMemcpyDeviceToHost);
	assert(
			utils::write_file(
					out_filename,
					h_out
					) == 0
			);

	hipFree(d_out);
	hipFree(d_in);
	free(h_out);
    return 0;
}
