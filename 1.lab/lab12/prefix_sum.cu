/*
 * Do not change this file
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cassert>
#include <cstring>
#include <string>
#include <chrono>
#include <cstdlib>
#include <ctime>
#define MAX_LENGTH 1024

/**
 * Read file, save edges to array (x_x) and 
 * record the size of each type of edge array (x_x_count).
 */
namespace utils {
	int target;
	int len;
	int *list;

	int read_file(std::string filename) {
		std::ifstream inputf(filename, std::ifstream::in);
		len = 0;
		list = (int*)malloc(sizeof(int) * MAX_LENGTH);
		if(inputf) {
			while (!inputf.eof()){
				inputf >> list[len];
				len++;
			}
		} else {
			return -1;
		}
		inputf.close();
		return 0;
	}

	int write_file(std::string filename, int *out){
		std::ofstream outputf(filename, std::ofstream::out);
		if(outputf.is_open()){
			for(int i = 0; i < len; i++){
				outputf << out[i] << std::endl;
			}
		} else {
			return -1;
		}
		outputf.close();
		return 0;
	}
}

/**
 * Global function: prefix sum
 * d_in:              original array
 * d_out:             prefix sum array (need to be allocated before)
 * numElems:          the sum of the array.
 * target_n:	      the target to check frequency.
 */
__global__ void prefix_sum_kernel(int *d_out, int *d_in, int numElems) {
	unsigned int d_hist_idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (d_hist_idx >= numElems)
	{
		return;
	}

	unsigned int cdf_val = 0;
	for (int i = 0; i <= d_hist_idx; ++i)
	{
		cdf_val = cdf_val + d_in[i];
	}
	d_out[d_hist_idx] = cdf_val;
}

__global__ void improved_prefix_sum_kernel(int *out, int *in, int n)
{
	__shared__ int temp[2049];

	int threadId = threadIdx.x;
	int offset = 1;

	//load input into shared memory
	temp[2 * threadId] = in[2 * threadId];
	temp[2 * threadId + 1] = in[2 * threadId + 1];
	__syncthreads();

	for(int d = n/2; d > 0; d /= 2) // build sum in place up the tree
	{
		__syncthreads();
		if(threadId < d)
		{
			int ai = offset * (2 * threadId + 1) - 1;
			int bi = offset * (2 * threadId + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if(threadId == 0) // clear the last element
		temp[n-1] = 0;

	for(int d = 1; d < n; d *= 2)
	{
		offset /= 2;
		__syncthreads();

		if(threadId < d)
		{
			int ai = offset * (2 * threadId + 1) - 1;
			int bi = offset * (2 * threadId + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	out[2 * threadId] = temp[2 * threadId + 1];
	out[2 * threadId + 1] = temp[2 * threadId + 2];
	if (threadId == 0) {
		out[n - 1] = out[n - 2] + in[n - 1];
	}
}


int main(int argc, char **argv) {
    assert(argc == 2 && "Input format error!");
    std::string filename = argv[1];
    assert(utils::read_file(
        	filename
    		) == 0
		);
	int numElems = utils::len;
	int *d_in;
	int *d_out;

	hipMalloc((void**)&d_in, sizeof(int) * numElems);
	hipMalloc((void**)&d_out, sizeof(int) * numElems);

	int *h_out = (int*)malloc(sizeof(int) * numElems);

	hipMemcpy(d_in, utils::list, sizeof(int) * numElems, hipMemcpyHostToDevice);

	dim3 grid(1);
	dim3 block(1024);

    auto t_start = std::chrono::high_resolution_clock::now();

    hipEvent_t cuda_start, cuda_end;
    hipEventCreate(&cuda_start);
    hipEventCreate(&cuda_end);
    float naive_kernel_time;
    float improved_kernel_time;
    hipEventRecord(cuda_start);

	prefix_sum_kernel<<<grid, block>>>(
			d_out,
			d_in,
			numElems);

    hipEventRecord(cuda_end);

    hipEventSynchronize(cuda_start);
    hipEventSynchronize(cuda_end);
    hipEventElapsedTime(&naive_kernel_time, cuda_start, cuda_end);
	hipDeviceSynchronize();
	hipMemcpy(h_out, d_out, sizeof(int) * numElems, hipMemcpyDeviceToHost);
	assert(
			utils::write_file(
					"naive_out.txt",
					h_out
			) == 0
	);

	hipEventRecord(cuda_start);

	improved_prefix_sum_kernel<<<grid, block>>>(
			d_out,
			d_in,
			numElems
			);

	hipEventRecord(cuda_end);

	hipEventSynchronize(cuda_start);
	hipEventSynchronize(cuda_end);
	hipEventElapsedTime(&improved_kernel_time, cuda_start, cuda_end);
	hipDeviceSynchronize();

	hipMemcpy(h_out, d_out, sizeof(int) * numElems, hipMemcpyDeviceToHost);
	assert(
			utils::write_file(
					"improved_out.txt",
					h_out
			) == 0
	);

    auto t_end = std::chrono::high_resolution_clock::now();


	fprintf(stderr, "Naive    prefix_sum Time: %.9lf s\n", naive_kernel_time / pow(10, 3));
	fprintf(stderr, "Improved Prefix_sum Time: %.9lf s\n", improved_kernel_time / pow(10, 3));

	hipFree(d_out);
	hipFree(d_in);
	free(h_out);
    return 0;
}