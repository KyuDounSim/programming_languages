
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

__global__ void common_elements(int* d_A, int* d_B, int* d_C) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int num_threads = blockDim.x * gridDim.x;

	d_C[tid] = 0;
	
	// loop over A and B and count the number of common elements
	// add your code here

	for (int i = tid; i < 2048; i += num_threads) {
		for (int j = 0; j < 2048; ++j){
			if (d_A[i] == d_B[j]){
				++d_C[tid];
			}
		}
    }
}

int main() {
	int *A = (int*)malloc(2048 * sizeof(int));
	int *B = (int*)malloc(2048 * sizeof(int));

	// read files
	std::ifstream inputa("a.txt", std::ifstream::in);
	std::ifstream inputb("b.txt", std::ifstream::in);
	for (int i = 0; i < 2048; i++) {
		inputa >> A[i];
		inputb >> B[i];
	}

	int num_blocks_per_grid = 4;
	int num_threads_per_grid = 32;

	// Allocate the memory in GPU to store the content of A,B,C
	int *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, 2048 * sizeof(int));
	hipMalloc((void **)&d_B, 2048 * sizeof(int));

	// d_C stores the number of common elements found by each thread
	hipMalloc((void **)&d_C, num_blocks_per_grid * num_threads_per_grid * sizeof(int));

	// Copy A, B to d_A,d_B
	hipMemcpy(d_A, A, 2048 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, 2048 * sizeof(int), hipMemcpyHostToDevice);

	common_elements<<<num_blocks_per_grid,num_threads_per_grid>>>(d_A, d_B, d_C);

	int *C = (int*)malloc(num_blocks_per_grid * num_threads_per_grid * sizeof(int));
	hipMemcpy(C, d_C, num_blocks_per_grid * num_threads_per_grid* sizeof(int), hipMemcpyDeviceToHost);
	
	int num_common_elements = 0;

    for (int i = 0; i < num_blocks_per_grid * num_threads_per_grid; i++) {
		num_common_elements += C[i];
	}

	// print the number of common elements
	std::cout << num_common_elements << std::endl;

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(A);
	free(B);
	free(C);

	return 0;
}
