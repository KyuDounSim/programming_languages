#include "hip/hip_runtime.h"
#include <iostream>
#include "helpers.h"
using namespace std;
/* you can define data structures and helper functions here */


/**
 * please remember to set final_results and final_result_size 
 * before return.
 */




/*
Input: n-o array
Output: array of Nitrogen id's that have two Oxygen attached.

__global__ void find_no2(int *d_out, int *d_in, int numElems) {
    unsigned int d_hist_idx = blockDim.x * blockIdx.x + threadIdx.x;

    cout << d_hist_idx << endl;
    if (d_hist_idx >= numElems)
    {
        return;
    }

    unsigned int cdf_val = 0;
    for (int i = 0; i <= d_hist_idx; ++i)
    {
        cdf_val = cdf_val + d_in[i];
    }
    d_out[d_hist_idx] = cdf_val;
}
*/


/*
Input: C-C array, C-C array length
Output: array of c's id that form c6 rings

1. Yes, because in that way, we need to first check the index of 1,3,5,7 which is an inefficient jump access.
2. Given an edge (x1,x2), you can explore all edges in the edge array to find the edges that starts with x2, say(x2,x3) and (x2, x4). Then you can get two larger structure{x1,x2,x3} and {x1,x2,x4}. After you repeat this process 6 times, you would get a c6ring.
*/

__global__ void find_c6ring(int *d_out, int *d_in, int c_c_size) {
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        int num_threads = blockDim.x + gridDim.x;

        int actual_c_num = c_c_size / 2;
        int cnt = 0;
        

        for(int i = tid; i < c_c_size; i += num_threads) {
            cnt = 0; 
            printf("Current elem is %d\n", d_in[i]); 
            for(int j = 0; j < c_c_size; ++j) {
                if(d_in[(i + actual_c_num) % c_c_size] == d_in[j]) {
                    ++cnt;
                    printf("As %d matches with %d and the connected edge is %d\n", d_in[i], d_in[j], d_in[(i + actual_c_num) % c_c_size]);
                }
            }
            printf("%d matches %d times\n", d_in[i], cnt); 
        }
        //printf("%d\n", cnt);
}


/*
Input: N-O array, N-O array length
Output: array of N's ids that have two Oxygen bonds
*/
__global__ void valid_no2(int *d_out, int *d_int, int n_o_size) {
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        int num_threads = blockDim.x + gridDim.x;

        for(int i = tid ; i < n_o_size; i += num_threads) {
             
        }  
}

void tnt_counting(int num_blocks_per_grid, int num_threads_per_block,
        int* c_c, int* c_n, int* c_h, int* n_o,
        int c_c_size, int c_n_size, int c_h_size, int n_o_size,
        int* &final_results, int &final_result_size) {

        //int total_elem_num = num_blocks_per_grid * num_threads_per_block;
        cout << c_c_size << endl;

        int *d_c_c, *d_c6_rings;
 
        /*
        for(int i = 0; i < c_n_size; ++i) {
            cout << c_n[i] << " ";
        }

        cout << endl;
        cout << c_n_size << endl;
        */
        //int *d_c_c_size; 
        //*d_c_n, *d_c_h, *d_n_o;
        hipMalloc((void **)&d_c_c, c_c_size * sizeof(int));
        hipMalloc((void **)&d_c6_rings, c_c_size * sizeof(int)); 
        //hipMalloc((void **)&d_c_c_size, sizeof(int));
        //cout << c_c_size << endl; 
        hipMemcpy(d_c_c, c_c, c_c_size * sizeof(int), hipMemcpyHostToDevice);
        //hipMemcpy(d_c_c_size, c_c_size, sizeof(int), hipMemcpyHostToDevice);
        find_c6ring<<<num_blocks_per_grid, num_threads_per_block>>>(d_c6_rings, d_c_c, c_c_size);

        int *h_c6_rings = (int*)malloc(c_c_size * sizeof(int));
        hipMemcpy(h_c6_rings, d_c6_rings, c_c_size * sizeof(int), hipMemcpyDeviceToHost);

        int *d_n_o, *d_valid_no2;

        hipMalloc((void **) &d_n_o, n_o_size * sizeof(int));
        hipMalloc((void **) &d_valid_no2, n_o_size * sizeof(int));
        //hipMemcpy(d_n_o, n_o, n_o_size * sizeof(int), hipMemcpyHostToDevice);
        //valid_no2<<<num_blocks_per_grid, num_threads_per_block>>>(d_valid_no2, d_n_o, n_o_size * sizeof(int), hipMemcpyHostToDevice);

        //int* h_valid_no2 = (int*)malloc(n_o_size * sizeof(int));
        //hipMemcpy(h_valid_no2, d_valid_no2, n_o_size * sizeof(int), hipMemcpyHostToDevice);  

        hipFree(d_c_c);  
        hipFree(d_c6_rings);
        hipFree(d_n_o);
        //hipFree(d_valid_no2); 
        free(h_c6_rings);
        //free(h_valid_no2);
} 
