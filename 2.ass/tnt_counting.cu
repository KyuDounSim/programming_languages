#include "hip/hip_runtime.h"
#include <iostream>
#include "helpers.h"
using namespace std;
/* you can define data structures and helper functions here */

/**
 * please remember to set final_results and final_result_size 
 * before return.
 */


/*
Input: C-C array, C-C array length
Output: array of c's id that form c6 rings

1. Yes, because in that way, we need to first check the index of 1,3,5,7 which is an inefficient jump access.

2. Given an edge (x1,x2), you can explore all edges in the edge array to find the edges that starts with x2, say(x2,x3) and (x2, x4). Then you can get two larger structure{x1,x2,x3} and {x1,x2,x4}. After you repeat this process 6 times, you would get a c6ring.

3. After thread 1 finds (x2, x3) and (x2, x4), we record the number 2, indicating that there are 2 results found be thread 1. Other threads may also record the number of results. Then we sum them together and pass this value to Host. Then Host allocate the memory based on the total number of the results. Then all GPU threads do the repeat to do the finding one more time. But this time they write the results they found to the allocated memory.
*/

__global__ void next_depth_len(int *d_in, int arr_len, bool* d_mask, int* d_out) {
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        int num_threads = blockDim.x + gridDim.x;

        int actual_c_num = arr_len / 2, cnt = 0;
        d_out[tid] = 0;
        for(int i = tid; i < actual_c_num; i += num_threads) {
            cnt = 0;
            //printf("Current elem is %d\n", d_in[i]); 
            for(int j = 0; j < arr_len; ++j) {
                if(!d_mask[i]) {
                    if(d_in[(i + actual_c_num) % arr_len] == d_in[(j + actual_c_num) % arr_len]) {
                        ++cnt;
                        //printf("As %d matches with %d and the connected edge is %d\n", d_in[i], d_in[j], d_in[(i + actual_c_num) % c_c_size]);
                        d_mask[i] = true;
                    }
                }
                
                // printf("%d matches %d times\n", d_in[i], cnt); 
            }

            printf("cnt is %d\n", cnt);
            d_out[tid] += cnt;
        }
        //printf("%d\n", cnt);
}

/*
Input: boolean mask for NO2, N-O Array, N-O Array Size 
Output: number of valid Nitrogens
*/
__global__ void valid_n_len(int *d_in, bool* d_no2_mask, int n_o_size, int& d_out) {
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        int num_threads = blockDim.x + gridDim.x;

        int cnt = 0, total = 0;
        for(int i = tid; i < n_o_size; i += num_threads) {
            cnt = 0;
            d_no2_mask[i] = true;
            for(int j = tid + 1; j < n_o_size; ++j) {
                if(!d_no2_mask[i]) { 
                    if(d_in[i] == d_in[j]) {
                        ++cnt;
                        d_no2_mask[j] = true;
                    }
                }
            }
 
            if(cnt == 1) {
                ++total; 
            }
        }
        
        d_out = total;
}


/*
Input: N-O array, N-O array length
Output: array of N's ids that have two Oxygen bonds
*/

__global__ void valid_no2(int *d_in, bool* d_no2_mask, int n_o_size, int* d_out) {
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        int num_threads = blockDim.x + gridDim.x;

        int cnt = 0;
        for(int i = tid; i < n_o_size; i += num_threads) {
            printf("tid is %d and the element is %d\n", i, d_in[i]); 
            cnt = 0;
            for(int j = tid + 1; j < n_o_size; ++j) {
                if(!d_no2_mask[i]) { 
                    if(d_in[i] == d_in[j]) {
                        ++cnt;
                        d_no2_mask[j] = true;
                    }
                }
            }
 
            if(cnt == 1) {
                d_out[tid] = d_in[i];
            }
        }  
}

void tnt_counting(int num_blocks_per_grid, int num_threads_per_block,
        int* c_c, int* c_n, int* c_h, int* n_o,
        int c_c_size, int c_n_size, int c_h_size, int n_o_size,
        int* &final_results, int &final_result_size) {

        // Find c6 rings
        int *d_c_c, *d_next_depth_len_arr;
        bool *d_mask, *h_mask = (bool*)malloc((c_c_size / 2) * sizeof(bool));

        for(int i = 0; i < c_c_size / 2; ++i) {
            h_mask[i] = false;
        }

        hipMalloc((void **)&d_c_c, c_c_size * sizeof(int));
        hipMalloc((void **)&d_next_depth_len_arr, (c_c_size / 2) * sizeof(int));
        hipMalloc((void **)&d_mask, (c_c_size / 2) * sizeof(bool));         
        hipMemcpy(d_c_c, c_c, c_c_size * sizeof(int), hipMemcpyHostToDevice); 
        hipMemcpy(d_mask, h_mask, (c_c_size / 2) * sizeof(bool), hipMemcpyHostToDevice);
        
        next_depth_len<<<num_blocks_per_grid, num_threads_per_block>>>(d_c_c, c_c_size, d_mask, d_next_depth_len_arr);

        int *h_next_depth_len_arr = (int*)malloc((c_c_size / 2) * sizeof(int));

        hipMemcpy(h_next_depth_len_arr, d_next_depth_len_arr, (c_c_size / 2) * sizeof(int), hipMemcpyDeviceToHost);
        for(int i = 0; i < c_c_size / 2; ++i)
        {
            cout << h_next_depth_len_arr[i] << " ";
        }

        cout << endl;
 
        int next_depth_total_len = 0; 
        for(int i = 0; i < c_c_size / 2; ++i)
        {
            next_depth_total_len += h_next_depth_len_arr[i];
        }

        int *d_1st_level;
        hipMalloc((void**) &d_1st_level, next_depth_total_len * sizeof(int));

        /*
        // Find valid Nitrogens with 2 oxygens 
        
        // First, find the number of valid n
        int *d_n_o, *d_valid_no2;
        int d_n_len;
        bool *d_no2_mask;
        hipMalloc((void**) &d_n_len, sizeof(int));
        hipMalloc((void **) &d_n_o, n_o_size * sizeof(int));
        hipMalloc((void **) &d_valid_no2, n_o_size * sizeof(int));
        hipMalloc((void **) &d_no2_mask, n_o_size * sizeof(bool)); 

        hipMemcpy(d_n_o, n_o, n_o_size * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_no2_mask, h_no2_mask, n_o_size * sizeof(bool), hipMemcpyHostToDevice); 
        

        bool *h_no2_mask = (bool*)malloc(n_o_size * sizeof(bool));
        for(int i = 0; i < n_o_size; ++i) {
            h_no2_mask[i] = false;
        }
        
        int* d_n_len; int h_n_len = 0;
        hipMalloc((void **) &d_n_len, sizeof(int)); 
        valid_n_len<<<num_blocks_per_grid, num_threads_per_block>>>(d_n_o, d_no2_mask, n_o_size, d_n_len);
        hipMemcpy(h_n_len, d_n_len, sizeof(int), hipMemcpyDeviceToHost);

        valid_n_len<<<num_blocks_per_grid, num_threads_per_block>>>(d_n_o, d_no2_mask, n_o_size, d_n_len);
        
        cout << h_n_len<< endl;

        // 2nd, construct the no2 array
        for(int i = 0; i < n_o_size; ++i) {
            h_no2_mask[i] = false;
        }
        
        int* h_valid_no2 = (int*)malloc(n_o_size * sizeof(int));
        hipMemcpy(h_valid_no2, d_valid_no2, n_o_size * sizeof(int), hipMemcpyDeviceToHost);  
        //valid_no2<<<num_blocks_per_grid, num_threads_per_block>>>(d_valid_no2, d_n_o, d_no2_mask, n_o_size);
        */
        
        // Find if alternating c is connected to a valid n 
        // Free all dynamic variables
        hipFree(d_c_c);  
        hipFree(d_next_depth_len_arr);
        //hipFree(d_n_o);
        //hipFree(d_valid_no2); 
        free(h_next_depth_len_arr);
        //free(h_valid_no2);
} 
