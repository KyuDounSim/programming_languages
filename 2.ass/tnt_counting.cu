#include "hip/hip_runtime.h"
#include <iostream>
#include "helpers.h"
using namespace std;
/* you can define data structures and helper functions here */


/**
 * please remember to set final_results and final_result_size 
 * before return.
 */


/*
Input: C-C array, C-C array length
Output: C-C array with c6 rings, C6ring array length
*/

__global__
void find_c6ring(int *d_out, int *d_in, int& c_c_len) {
    unsigned int d_hist_idx = blockDim.x * blockIdx.x + threadIdx.x;

    cout << d_hist_idx << endl;
    if (d_hist_idx >= numElems)
    {
        return;
    }

    unsigned int cdf_val = 0;
    for (int i = 0; i <= d_hist_idx; ++i)
    {
        cdf_val = cdf_val + d_in[i];
    }
    d_out[d_hist_idx] = cdf_val;
}

/*
Input: n-o array
Output: array of Nitrogen id's that have two Oxygen attached.

__global__ void find_no2(int *d_out, int *d_in, int numElems) {
    unsigned int d_hist_idx = blockDim.x * blockIdx.x + threadIdx.x;

    cout << d_hist_idx << endl;
    if (d_hist_idx >= numElems)
    {
        return;
    }

    unsigned int cdf_val = 0;
    for (int i = 0; i <= d_hist_idx; ++i)
    {
        cdf_val = cdf_val + d_in[i];
    }
    d_out[d_hist_idx] = cdf_val;
}
*/

void tnt_counting(int num_blocks_per_grid, int num_threads_per_block,
        int* c_c, int* c_n, int* c_h, int* n_o,
        int c_c_size, int c_n_size, int c_h_size, int n_o_size,
        int* &final_results, int &final_result_size) {

        int* d_c_c, *d_c_n, *d_c_h, *d_n_o;
        
        hipMalloc((void **)&d_c_c, 2048 * sizeof(int));
        hipMalloc((void **)&d_c_n, 2048 * sizeof(int));
        hipMalloc((void **)&d_c_h, 2048 * sizeof(int));
        hipMalloc((void **)&d_n_o, 2048 * sizeof(int));

        find_c6ring<<<num_blocks_per_grid, num_threds_per_block>>>(c_c, c_c_size);  
        
        int* c6_rings = (int*)malloc(num_blocks_per_grid * num_threds_per_block * sizeof(int));
        hipMemcpy(c6_rings, d_c_c, numb_blocks_per_grid * num_threads_per_blck * sizeof(int), hipMemcpyDeviceToHost);

        hipFree(c6_rings);
        free(d_c_c);
        free(d_c_n);
        free(d_c_h);
        free(d_c_n);  
} 
