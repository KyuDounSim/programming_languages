#include "hip/hip_runtime.h"
#include <iostream>
#include "helpers.h"

/* you can define data structures and helper functions here */


__global__ void unique_c_mask(int *d_c_c, int arr_len, bool* d_visited, bool *d_result_mask) {
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        int num_threads = blockDim.x + gridDim.x;

        for(int i = tid; i < arr_len; i += num_threads) {
            for(int j = 0; j < arr_len; ++j) {
                d_result_mask[j] = true;
                if(d_c_c[i] == d_c_c[j]) {
                    d_result_mask[j] = false;
                }

                /* 
                if(!d_visited[j]) {
                    d_visited[j] = true;
                    if(d_c_c[i] != d_c_c[j]) {
                        printf("Diff %d vs %d\n", d_c_c[i], d_c_c[j]);
                        d_result_mask[j] = true;
                    } else {
                        d_result_mask[j] = false;
                    }
                } 
                */
            }
        }
}

__global__ void createRing(int* d_cc, int cc_len, int* d_cn, int cn_len, int* d_out_ring) {
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        int num_threads = blockDim.x + gridDim.x;
        int cc_len_half = cc_len / 2;

        for(int i = tid; i < cn_len; i += num_threads) {
            for(int j = i + 1; j < cn_len; ++j) {
                for(int k = j + 1; k < cn_len; ++k) {
                    int a = d_cn[i], b = d_cn[j], c = d_cn[k];
                    int a_b_cnt = 0, b_c_cnt = 0, c_a_cnt = 0; 
                    for(int l = 0; l < cc_len; ++l) {
                        if(d_cc[l] == a || d_cc[l] == b || d_cc[l] == c) {
                            continue;
                        }

                        for(int m = l + 1; m < cc_len; ++m) {
                            if(d_cc[m] == a || d_cc[m] == b || d_cc[m] == c) {
                                continue;
                            }

                            if(d_cc[l] == d_cc[m]) {
                                if(d_cc[(l + cc_len_half) % cc_len] == a && d_cc[(m + cc_len_half) % cc_len] == b || d_cc[(l + cc_len_half) % cc_len] == b && d_cc[(m + cc_len_half) % cc_len] == a) {
                                    printf("a_b_cnt is going up by %d and %d\n", l, m);
                                    ++a_b_cnt;
                                }
 
                                if(d_cc[(l + cc_len_half) % cc_len] == b && d_cc[(m + cc_len_half) % cc_len] == c || d_cc[(l + cc_len_half) % cc_len] == c && d_cc[(m + cc_len_half) % cc_len] == b) {
                                    printf("b_c_cnt is going up by %d and %d\n", l, m);
                                    ++b_c_cnt;
                                }
                            
                                if(d_cc[(l + cc_len_half) % cc_len] == c && d_cc[(m + cc_len_half) % cc_len] == a || d_cc[(l + cc_len_half) % cc_len] == a && d_cc[(m + cc_len_half) % cc_len] == c) {
                                    printf("c_a_cnt is going up by %d and %d\n", d_cc[l], d_cc[m]);
                                    ++c_a_cnt;
                                }
                            }
                        }
                    }
                     
                    int total = a_b_cnt * b_c_cnt * c_a_cnt;
                    
                    if(total == 0) {
                        continue;
                    }

                    d_out_ring[tid] = total;   
                    //printf("a_b_cnt is %d\n", a_b_cnt); printf("b_c_cnt is %d\n", b_c_cnt); printf("c_a_cnt is %d\n", c_a_cnt);
                } 
            }
        }
}

__global__ void createRing_fin(int* d_cc, int cc_len, int* d_cn, int cn_len, int* d_out_ring) {
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        int num_threads = blockDim.x + gridDim.x;
        int cc_len_half = cc_len / 2;

        for(int i = tid; i < cn_len; i += num_threads) {
            for(int j = i + 1; j < cn_len; ++j) {
                for(int k = j + 1; k < cn_len; ++k) {
                    int a = d_cn[i], b = d_cn[j], c = d_cn[k];
                    int a_b_cnt = 0, b_c_cnt = 0, c_a_cnt = 0; 
                    for(int l = 0; l < cc_len; ++l) {
                        if(d_cc[l] == a || d_cc[l] == b || d_cc[l] == c) {
                            continue;
                        }

                        for(int m = l + 1; m < cc_len; ++m) {
                            if(d_cc[m] == a || d_cc[m] == b || d_cc[m] == c) {
                                continue;
                            }

                            if(d_cc[l] == d_cc[m]) {
                                if(d_cc[(l + cc_len_half) % cc_len] == a && d_cc[(m + cc_len_half) % cc_len] == b || d_cc[(l + cc_len_half) % cc_len] == b && d_cc[(m + cc_len_half) % cc_len] == a) {
                                    printf("a_b_cnt is going up by %d and %d\n", l, m);
                                    ++a_b_cnt;
                                }
 
                                if(d_cc[(l + cc_len_half) % cc_len] == b && d_cc[(m + cc_len_half) % cc_len] == c || d_cc[(l + cc_len_half) % cc_len] == c && d_cc[(m + cc_len_half) % cc_len] == b) {
                                    printf("b_c_cnt is going up by %d and %d\n", l, m);
                                    ++b_c_cnt;
                                }
                            
                                if(d_cc[(l + cc_len_half) % cc_len] == c && d_cc[(m + cc_len_half) % cc_len] == a || d_cc[(l + cc_len_half) % cc_len] == a && d_cc[(m + cc_len_half) % cc_len] == c) {
                                    printf("c_a_cnt is going up by %d and %d\n", d_cc[l], d_cc[m]);
                                    ++c_a_cnt;
                                }
                            }
                        }
                    }
                     
                    int total = a_b_cnt * b_c_cnt * c_a_cnt;
                    
                    if(total == 0) {
                        continue;
                    }

                    d_out_ring[tid] = total;   
                    //printf("a_b_cnt is %d\n", a_b_cnt); printf("b_c_cnt is %d\n", b_c_cnt); printf("c_a_cnt is %d\n", c_a_cnt);
                } 
            }
        }
}


/*
Input : n-o array 
Output: id of n with 2
*/
__global__ void valid_n(int* d_no, int no_len, int* d_valid_no, int* d_valid_no_len) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int num_threads = blockDim.x + gridDim.x;

    int valid_no_len = 0;
    int only_c = no_len / 2, cnt = 0;
    for(int i = tid; i < no_len; i += num_threads) {
        cnt = 0; 
        for(int j = 0; j < no_len; ++j) {
            if(d_no[i] == d_no[j]) {
                ++cnt;
                printf("%d has 2 counts\n", d_no[i]);
            }
        }
        
        if(cnt == 2) {
            printf("%d has 2 counts\n", d_no[i]);
            d_valid_no[i] = d_no[i];
            ++valid_no_len; 
        }
    }
    
    d_valid_no_len[0] = valid_no_len;
}
/**
 * please remember to set final_results and final_result_size 
 * before return.
 */
void tnt_counting(int num_blocks_per_grid, int num_threads_per_block,
        int* c_c, int* c_n, int* c_h, int* n_o,
        int c_c_size, int c_n_size, int c_h_size, int n_o_size,
        int* &final_results, int &final_result_size) {
            std::cout << n_o_size << std::endl; 
            int* d_no, *d_valid_no, *d_valid_no_len;
 
            hipMalloc((void**) &d_no, n_o_size * sizeof(int));
            hipMalloc((void**) &d_valid_no, n_o_size * sizeof(int));
            hipMalloc((void**) &d_valid_no_len, sizeof(int));
            
            hipMemcpy(d_no, n_o, n_o_size * sizeof(int), hipMemcpyHostToDevice);
            valid_n<<<num_blocks_per_grid, num_threads_per_block>>>(d_no, n_o_size, d_valid_no, d_valid_no_len); 

            int *h_valid_no = (int*) malloc(n_o_size * sizeof(int)), *h_valid_no_len = (int*) malloc(sizeof(int));
            hipMemcpy(h_valid_no, d_valid_no, n_o_size * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(h_valid_no_len, d_valid_no_len, sizeof(int), hipMemcpyDeviceToHost);

            std::cout << "valid_no2_len is " << h_valid_no_len[0] << std::endl;

            for(int i = 0; i < h_valid_no_len[0]; ++i)
                std::cout << h_valid_no[i] << " ";
            
            std::cout << std::endl;

            int* h_valid_c = (int*) malloc(3 * sizeof(int));
            h_valid_c[0] = 0; h_valid_c[1] = 2; h_valid_c[2] = 4;

            int* d_valid_c;
            hipMalloc((void **) &d_valid_c, 3 * sizeof(int));
            hipMemcpy(d_valid_c, h_valid_c, 3 * sizeof(int), hipMemcpyHostToDevice);

            int *d_c_c;
            hipMalloc((void**) &d_c_c, c_c_size * sizeof(int)); 
            hipMemcpy(d_c_c, c_c, c_c_size * sizeof(int), hipMemcpyHostToDevice);
            
            int *d_ring;
            hipMalloc((void**) &d_ring, sizeof(int));
            int *h_ring = (int*) malloc(sizeof(int));
            createRing<<<num_blocks_per_grid, num_threads_per_block>>>(d_c_c, c_c_size, d_valid_c, 3, d_ring); 
            hipMemcpy(h_ring, d_ring, sizeof(int), hipMemcpyDeviceToHost);

            std::cout << h_ring[0] << std::endl;

            int *d_rings;
            hipMalloc((void**) &d_rings, h_ring[0] * 6 * sizeof(int));

            int *h_rings = (int*) malloc(h_ring[0] * 6 * sizeof(int));
            
            createRing_fin<<<num_blocks_per_grid, num_threads_per_block>>>(d_c_c, c_c_size, d_valid_c, 3, d_rings);
            hipMemcpy(h_rings, d_rings, h_ring[0] * 6 * sizeof(int), hipMemcpyDeviceToHost);
            
            for(int i = 0; i < 2; ++i) {
                for(int j = 0; j < 6; ++j) {
                    std::cout << h_rings[j] << " "; 
                }
                std::cout << std::endl;
            } 
            /*// maksing unique c starting points
            int * d_c_c; 
            hipMalloc((void **) &d_c_c, c_c_size * sizeof(int)); 
            
            bool *h_unique_c_mask, *h_c_visited;
            h_unique_c_mask = (bool*) malloc((c_c_size / 2) * sizeof(bool));
            h_c_visited = (bool*) malloc((c_c_size / 2) * sizeof(bool));
            
            for(int i = 0; i < c_c_size / 2; ++i) {
                h_c_visited[i] = false;
            }

            bool* d_unique_c_mask, *d_c_visited; 
            hipMalloc((void **) &d_unique_c_mask, c_c_size / 2 * sizeof(bool));
            hipMalloc((void **) &d_c_visited, c_c_size / 2 * sizeof(bool));

            hipMemcpy(d_c_c, c_c, c_c_size * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_unique_c_mask, h_unique_c_mask, (c_c_size / 2 ) * sizeof(bool), hipMemcpyHostToDevice);
            hipMemcpy(d_c_visited, h_c_visited, (c_c_size / 2 ) * sizeof(bool), hipMemcpyHostToDevice);

            unique_c_mask<<<num_blocks_per_grid, num_threads_per_block>>>(d_c_c, c_c_size / 2, d_c_visited, d_unique_c_mask); 
            hipMemcpy(h_unique_c_mask, d_unique_c_mask, (c_c_size / 2) * sizeof(bool), hipMemcpyDeviceToHost);

            for(int i = 0; i < c_c_size / 2; ++i) {
                std::cout << h_unique_c_mask[i] << std::endl; 
            }
            
            hipFree(d_c_c);
            hipFree(d_unique_c_mask);
            hipFree(d_c_visited);
            free(h_unique_c_mask);
            free(h_c_visited);*/
}
