#include "hip/hip_runtime.h"
#include <iostream>
#include "helpers.h"

/* you can define data structures and helper functions here */


/*
__global__ void unique_c_mask(int *d_c_c, int arr_len, bool* d_visited, bool *d_result_mask) {
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        int num_threads = blockDim.x + gridDim.x;

        for(int i = tid; i < arr_len; i += num_threads) {
            for(int j = tid + 1; j < arr_len; ++j) {
                if(!d_visited[j]) {
                    d_visited[j] = true;
                    if(d_c_c[i] != d_c_c[j]) {
                        printf("Diff %d vs %d\n", d_c_c[i], d_c_c[j]);
                        d_result_mask[j] = true;
                    } else {
                        d_result_mask[j] = false;
                    }
                } 
            }
        }
}
*/


/**
 * please remember to set final_results and final_result_size 
 * before return.
 */
void tnt_counting(int num_blocks_per_grid, int num_threads_per_block,
        int* c_c, int* c_n, int* c_h, int* n_o,
        int c_c_size, int c_n_size, int c_h_size, int n_o_size,
        int* &final_results, int &final_result_size) {

            /* // maksing unique c starting points
            int * d_c_c; 
            hipMalloc((void **) &d_c_c, c_c_size * sizeof(int)); 
            
            bool *h_unique_c_mask, *h_c_visited;
            h_unique_c_mask = (bool*) malloc((c_c_size / 2) * sizeof(bool));
            h_c_visited = (bool*) malloc((c_c_size / 2) * sizeof(bool));
            
            for(int i = 0; i < c_c_size / 2; ++i) {
                h_c_visited[i] = false;
            }

            bool* d_unique_c_mask, *d_c_visited; 
            hipMalloc((void **) &d_unique_c_mask, c_c_size / 2 * sizeof(bool));
            hipMalloc((void **) &d_c_visited, c_c_size / 2 * sizeof(bool));

            hipMemcpy(d_c_c, c_c, c_c_size * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_unique_c_mask, h_unique_c_mask, (c_c_size / 2 ) * sizeof(bool), hipMemcpyHostToDevice);
            hipMemcpy(d_c_visited, h_c_visited, (c_c_size / 2 ) * sizeof(bool), hipMemcpyHostToDevice);

            unique_c_mask<<<num_blocks_per_grid, num_threads_per_block>>>(d_c_c, c_c_size / 2, d_c_visited, d_unique_c_mask); 
            hipMemcpy(h_unique_c_mask, d_unique_c_mask, (c_c_size / 2) * sizeof(bool), hipMemcpyDeviceToHost);

            for(int i = 0; i < c_c_size / 2; ++i) {
                std::cout << h_unique_c_mask[i] << std::endl; 
            }
            
            hipFree(d_c_c);
            hipFree(d_unique_c_mask);
            hipFree(d_c_visited);
            free(h_unique_c_mask);
            free(h_c_visited); */
            
}
