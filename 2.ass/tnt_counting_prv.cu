#include "hip/hip_runtime.h"
#include <iostream>
#include <setjmp.h>
#include "helpers.h"

/* you can define data structures and helper functions here */

__global__ void valid_n_counter(int* cn, int cn_len, int* no, int no_len, bool* valid_n_num)  {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    for(int i = tid; i < cn_len ; i += num_threads) {
        int n_ctr = 0;
        
        for(int j = 0; j < no_len; ++j) {
            if(cn[i + cn_len] == no[j]) {
                ++n_ctr;
            }
        }

        if(n_ctr == 2) {
            //printf("tid is %d", tid);
            valid_n_num[tid] = true;
        }
    }
}

__global__ void combination_generator(int * cn, bool* mask, int cn_len, int* d_out) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int num_threads = blockDim.x + gridDim.x;

    int total_num = 0;

    for(int i = tid; i < cn_len; i += num_threads) {
        for(int j = i + 1; j < cn_len; ++j) {
            for(int k = j + 1; k < cn_len; ++k) {
                if(mask[i] == true && mask[j] == true && mask[k] == true) {
                    //printf("%d and %d and %d\n", cn[i], cn[j], cn[k]);
                    d_out[total_num * 3 + 0] = cn[i];
                    d_out[total_num * 3 + 1] = cn[j];
                    d_out[total_num * 3 + 2] = cn[k];
                    ++total_num;
                }
            }
        }
    }
}

__global__ void find_connectors(int* d_cc, int cc_len, int* d_cn, int cn_len, int* d_out_ring) {
        int tid = blockDim.x * blockIdx.x + threadIdx.x;
        int num_threads = blockDim.x * gridDim.x;
        int cc_len_half = cc_len / 2;
        int ring_cnt = 0;

        for(int i = tid; i < cn_len; i += num_threads) {
            for(int j = i + 1; j < cn_len; ++j) {
                for(int k = j + 1; k < cn_len; ++k) {
                    int a = d_cn[i], b = d_cn[j], c = d_cn[k];
                    int a_b_cnt = 0, b_c_cnt = 0, c_a_cnt = 0;
                    for(int l = 0; l < cc_len; ++l) {
                        if(d_cc[l] == a || d_cc[l] == b || d_cc[l] == c) {
                            continue;
                        }
                        
                        for(int m = l + 1; m < cc_len; ++m) {
                            if(d_cc[m] == a || d_cc[m] == b || d_cc[m] == c) {
                                continue;
                            }

                            if(d_cc[l] == d_cc[m]) {
                                if(d_cc[(l + cc_len_half) % cc_len] == a && d_cc[(m + cc_len_half) % cc_len] == b || d_cc[(l + cc_len_half) % cc_len] == b && d_cc[(m + cc_len_half) % cc_len] == a) {
                                    //printf("a_b_cnt is going up by %d and %d\n", l, m);
                                    ++a_b_cnt;
                                }
 
                                if(d_cc[(l + cc_len_half) % cc_len] == b && d_cc[(m + cc_len_half) % cc_len] == c || d_cc[(l + cc_len_half) % cc_len] == c && d_cc[(m + cc_len_half) % cc_len] == b) {
                                    //printf("b_c_cnt is going up by %d and %d\n", l, m);
                                    ++b_c_cnt;
                                }
                            
                                if(d_cc[(l + cc_len_half) % cc_len] == c && d_cc[(m + cc_len_half) % cc_len] == a || d_cc[(l + cc_len_half) % cc_len] == a && d_cc[(m + cc_len_half) % cc_len] == c) {
                                    //printf("c_a_cnt is going up by %d and %d\n", d_cc[l], d_cc[m]);
                                    ++c_a_cnt;
                                }
                            }
                        }
                    }
                     
                    int total = a_b_cnt * b_c_cnt * c_a_cnt;
                    
                    if(total == 0) {
                        continue;
                    }

                    d_out_ring[ring_cnt * 3 + 0] = a_b_cnt;
                    d_out_ring[ring_cnt * 3 + 1] = b_c_cnt;
                    d_out_ring[ring_cnt * 3 + 2] = c_a_cnt;
                    ++ring_cnt;
                    //d_out_ring[tid] = total;   
                    //printf("a_b_cnt is %d\n", a_b_cnt); printf("b_c_cnt is %d\n", b_c_cnt); printf("c_a_cnt is %d\n", c_a_cnt);
                } 
            }
        }
}

//__global__ void createRing(int* d_cc, int cc_len, int* d_c_tuple, int tuple_len, int* d_connector_num, int d_connector_num_size, int total_ring_num, int* d_out_ring) {
__global__ void createRing(int* d_cc, int cc_len, int* d_c_tuple, int tuple_len, int* d_connector_num, int total_ring_num, int* d_out_ring) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    //int num_threads = blockDim.x * gridDim.x;

    int cc_len_half = cc_len / 2;

    int a = 0, b = 0, c = 0, a_b_cnt = 0, b_c_cnt = 0, c_a_cnt = 0;

    // tuple_len == 1
    for(int i = 0; i < tuple_len; i += 3) {
    //for(int i = tid; i < tuple_len; i += (3 * num_threads)) {
        //printf("what is i or tid? It is %d\n", i);
        a = d_c_tuple[i * 3 + 0]; b = d_c_tuple[i * 3 + 1]; c = d_c_tuple[i * 3 + 2];
        //printf("a, b, c is each %d, %d, %d\n", a, b, c);

        a_b_cnt = 0;
        b_c_cnt = 0;
        c_a_cnt = 0;
        
        for(int ringNum = 0; ringNum < total_ring_num; ++ringNum) {
            d_out_ring[6 * ringNum + 1] = a;
            d_out_ring[6 * ringNum + 3] = b;
            d_out_ring[6 * ringNum + 5] = c;
        }

        for(int j = tid; j < cc_len; ++j) {
            if(d_cc[j] == a || d_cc[j] == b || d_cc[j] == c) {
                continue;
            }

            for(int k = j + 1; k < cc_len; ++k) {
                if(d_cc[k] == a || d_cc[k] == b || d_cc[k] == c || j == k) {
                    continue;
                }

                if(d_cc[j] == d_cc[k]) {
                    if(d_cc[(j + cc_len_half) % cc_len] == a && d_cc[(k + cc_len_half) % cc_len] == b || d_cc[(j + cc_len_half) % cc_len] == b && d_cc[(k + cc_len_half) % cc_len] == a) {
                        int repeat = total_ring_num / d_connector_num[i * 3 + 0];
                        for(int z = 0; z < repeat; ++z) {
                            d_out_ring[6 * i + 6 * z + 6 * a_b_cnt + 0] = d_cc[k];
                        }
                        ++a_b_cnt;
                    }

                    if(d_cc[(j + cc_len_half) % cc_len] == b && d_cc[(k + cc_len_half) % cc_len] == c || d_cc[(j + cc_len_half) % cc_len] == c && d_cc[(k + cc_len_half) % cc_len] == b) {
                        int repeat = total_ring_num / d_connector_num[i * 3 + 1];
                        for(int z = 0; z < repeat; ++z) {
                            d_out_ring[6 * i + 6 * z + 6 * b_c_cnt + 2] = d_cc[k];
                        }
                        ++b_c_cnt;
                    }
                
                    if(d_cc[(j + cc_len_half) % cc_len] == c && d_cc[(k + cc_len_half) % cc_len] == a || d_cc[(j + cc_len_half) % cc_len] == a && d_cc[(k + cc_len_half) % cc_len] == c) {
                        int repeat = total_ring_num / d_connector_num[i * 3 + 2];
                        for(int z = 0; z < repeat; ++z) {
                            d_out_ring[6 * i + 6 * z + 6 * c_a_cnt + 4] = d_cc[k];
                        }
                        ++c_a_cnt;
                    }
                }
            }
        }
        
        /*
        for(int tup1 = 0; tup1 < d_connector_num[tup1]; ++tup1) {
            for(int tup2 = 0; tup2 < d_connector_num[tup2]; ++tup2) {
                for(int tup3 = 0; tup2 < d_connector_num[tup3]; ++tup3) {

                }
            }
        }
        */
        
        /*
        for(int search = 0; search < total_ring_num; ++search) {
            d_out_ring[6 * search + 1] = a;
            d_out_ring[6 * search + 3] = b;
            d_out_ring[6 * search + 5] = c;

            for(int j = tid; j < cc_len; j += num_threads) {
                if(d_cc[j] == a || d_cc[j] == b || d_cc[j] == c) {
                    continue;
                }

                //int a_b_store = -100, b_c_store = -100, int c_a_store = -100;
                //int total = 0;

                for(int k = j + 1; k < cc_len; ++k) {

                    if(d_cc[k] == a || d_cc[k] == b || d_cc[k] == c || j == k) {
                        continue;
                    }

                    if(d_cc[j] == d_cc[k]) {
                        if(d_cc[(j + cc_len_half) % cc_len] == a && d_cc[(k + cc_len_half) % cc_len] == b || d_cc[(j + cc_len_half) % cc_len] == b && d_cc[(k + cc_len_half) % cc_len] == a) {
                                //a_b_store = d_cc[k];
                                printf("a_b is %d", d_connector_num[j]);
                                d_out_ring[6 * search + 0] = d_cc[k];
                                ++a_b_cnt;
                        }

                        if(d_cc[(j + cc_len_half) % cc_len] == b && d_cc[(k + cc_len_half) % cc_len] == c || d_cc[(j + cc_len_half) % cc_len] == c && d_cc[(k + cc_len_half) % cc_len] == b) {
                                //b_c_store = d_cc[k];
                                d_out_ring[6 * search + 2] = d_cc[k];
                                ++b_c_cnt;
                        }
                    
                        if(d_cc[(j + cc_len_half) % cc_len] == c && d_cc[(k + cc_len_half) % cc_len] == a || d_cc[(j + cc_len_half) % cc_len] == a && d_cc[(k + cc_len_half) % cc_len] == c) {
                                //printf("Entry added with %dth ring in %d\n", search, d_cc[k]);
                                d_out_ring[6 * search + 4] = d_cc[k];
                                ++c_a_cnt;
                        }
                    }

                    //total = a_b_cnt * b_c_cnt * c_a_cnt;                
                    //if(total > 0) {
                    //    printf("Entry added with %dth ring\n", constructed_ring);
                    //    ++constructed_ring;
                    //}
                }
            }
        }
        */
    }
}

__global__ void addNO2(int* d_c6, int ring_num, int* d_cn, int cn_size, int* d_no, int no_size, int* d_out) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    for(int i = 0; i < ring_num; ++i) {
        for(int j = 0; j < 6; ++j) {
            d_out[15 * i + j] = d_c6[6 * i + j];

            int cn = d_c6[6 * i + j];
            if(j == 1) {
                for(int k = tid; k < cn_size; k += num_threads) {
                    if(cn == d_cn[k]) {
                        d_out[15 * i + 6] = d_cn[k + cn_size];
                        int cnt = 0, n = d_cn[k + cn_size];
                        for(int l = 0; l < no_size; ++l) {
                            if(n == d_no[l]) {
                                if(cnt == 0) {
                                    d_out[15 * i + 9 + cnt] = d_no[l + no_size];
                                    ++cnt;
                                } else {
                                    d_out[15 * i + 9 + cnt] = d_no[l + no_size];
                                }
                            }
                        }
                    }
                }
            } else if (j == 3) {
                for(int k = tid; k < cn_size; k += num_threads) {
                    if(cn == d_cn[k]) {
                        d_out[15 * i + 7] = d_cn[k + cn_size];
                        int cnt = 0, n = d_cn[k + cn_size];
                        for(int l = 0; l < no_size; ++l) {
                            if(n == d_no[l]) {
                                if(cnt == 0) {
                                    d_out[15 * i + 11 + cnt] = d_no[l + no_size];
                                    ++cnt;
                                } else {
                                    d_out[15 * i + 11 + cnt] = d_no[l + no_size];
                                }
                            }
                        }
                    }
                }
            } else if(j == 5) {
                for(int k = tid; k < cn_size; k += num_threads) {
                    if(cn == d_cn[k]) {
                        d_out[15 * i + 8] = d_cn[k + cn_size];
                        int cnt = 0, n = d_cn[k + cn_size];
                        for(int l = 0; l < no_size; ++l) {
                            if(n == d_no[l]) {
                                if(cnt == 0) {
                                    d_out[15 * i + 13 + cnt] = d_no[l + no_size];
                                    ++cnt;
                                } else {
                                    d_out[15 * i + 13 + cnt] = d_no[l + no_size];
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}

/*
__global__ void add48Mappings(int* d_tnt, int ring_num, int* d_out) {

    // (012345 678) (9 10) (11 12) (13 14)
    
    for(int ring = 0; ring < ring_num; ++ring) {
        int base = 15 * 48 * ring;

        for(int i = 0; i < 6; ++i) {
        
            d_out[base + i] = d_tnt[15 * ring_num + i];


            for(int j = 0; j < 2; ++j) {


                for(int k = 0; k < 2; ++k) {


                    for(int l = 0; l < 2; ++l) {



                    }
                }
            }
        }
    }
}
*/

/**
 * please remember to set final_results and final_result_size 
 * before return.
 */

void tnt_counting(int num_blocks_per_grid, int num_threads_per_block,
        int* c_c, int* c_n, int* c_h, int* n_o,
        int c_c_size, int c_n_size, int c_h_size, int n_o_size,
        int* &final_results, int &final_result_size) {

            // validate c with n02
            int* d_cn;
            bool *d_valid_cn_len;
            hipMalloc((void**) &d_cn, 2 * c_n_size * sizeof(int));
            hipMalloc((void**) &d_valid_cn_len, c_n_size * sizeof(bool));

            int* d_no, *d_valid_no;
            hipMalloc((void**) &d_no, 2 * n_o_size * sizeof(int));
            hipMalloc((void**) &d_valid_no, 2 * n_o_size * sizeof(int));

            hipMemcpy(d_no, n_o, 2 * n_o_size * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_cn, c_n, 2 * c_n_size * sizeof(int), hipMemcpyHostToDevice);
            
            valid_n_counter<<<num_blocks_per_grid, num_threads_per_block>>>(d_cn, c_n_size, d_no, n_o_size, d_valid_cn_len);
            hipDeviceSynchronize();

            bool * h_valid_cn = (bool*) malloc(c_n_size * sizeof(bool));
            hipMemcpy(h_valid_cn, d_valid_cn_len, c_n_size * sizeof(bool), hipMemcpyDeviceToHost);


            /*
            //1 1 1
            for(int i = 0; i < c_n_size; ++i) {
                std::cout << h_valid_cn[i] << " ";
            }
            std::cout << std::endl;
            */

            // 3 -> therefore only 1 combination possible
            int good_cno2_num = 0;
            for(int i = 0; i < c_n_size; ++i) {
                if(h_valid_cn[i] == true) {
                    ++good_cno2_num;
                }
            }

            // all possible # of combination;
            unsigned int comb_cnt = 1;
            for(int i = good_cno2_num; i > good_cno2_num - 3; --i) {
                comb_cnt *= i;
            }
            comb_cnt /= 6;

            // filter out and create tuple array

            int* d_3_tuple;
            hipMalloc((void**) &d_3_tuple, 3 * comb_cnt * sizeof(int));
            hipMemcpy(d_3_tuple, h_valid_cn, c_n_size * sizeof(int), hipMemcpyHostToDevice);

            combination_generator<<<num_blocks_per_grid, num_threads_per_block>>>(d_cn, d_valid_cn_len, c_n_size, d_3_tuple);
            hipDeviceSynchronize();

            int* h_3_tuple = (int*) malloc(3 * comb_cnt * sizeof(int));
            hipMemcpy(h_3_tuple, d_3_tuple, 3 * comb_cnt * sizeof(int), hipMemcpyDeviceToHost);



            // prints out 0 2 4
            /*
            for(int i = 0; i < comb_cnt; ++i) {
                for(int j = 0; j < 3; ++j) {
                    std::cout << h_3_tuple[i * 3 + j] << " ";
                }
                std::cout << std::endl;
            }
            */

            // c1 c2 c3 사이에 1, 1, 2 이거 알려주는거임
            int *d_c_c;
            hipMalloc((void**) &d_c_c, c_c_size * sizeof(int)); 
            hipMemcpy(d_c_c, c_c, c_c_size * sizeof(int), hipMemcpyHostToDevice);


            int *d_connector_num;
            hipMalloc((void**) &d_connector_num, 3 * comb_cnt * sizeof(int));

            find_connectors<<<num_blocks_per_grid, num_threads_per_block>>>(d_c_c, c_c_size, d_3_tuple, comb_cnt * 3, d_connector_num);
            hipDeviceSynchronize();

            int *h_connector_num = (int*) malloc(comb_cnt * 3 * sizeof(int));
            hipMemcpy(h_connector_num, d_connector_num, 3 * sizeof(int), hipMemcpyDeviceToHost);
            

            /*
            std::cout << "number of each connectors called createRing_fin" << std::endl;
            for(int i = 0; i < comb_cnt * 3 ; ++i) {
                std::cout << h_connector_num[i] << " ";
            }
            std::cout << std::endl;
            */

            int total_rings = 0;

            for(int i = 0; i < comb_cnt * 3; i += 3) {
                int temp = 1;
                for(int j = 0; j < 3; ++j) {
                    temp *= h_connector_num[i * 3 + j];
                }
                total_rings += temp;
            }

            //std::cout << "Total ring num is " << total_rings << std::endl;

            //hipMemcpy(d_connector_num, h_connector_num, 3 * sizeof(int), hipMemcpyHostToDevice);

            int *d_rings;
            hipMalloc((void**) &d_rings, total_rings * 6 * sizeof(int));

            int *h_rings = (int*) malloc(total_rings * 6 * sizeof(int));

            // int* d_cc, int cc_len, int* d_c_tuple, int tuple_len, int* d_connector_num,
            // int d_connector_num_size, int total_ring_num, int* d_out_ring

            createRing<<<num_blocks_per_grid, num_threads_per_block>>>(d_c_c, c_c_size, d_3_tuple, comb_cnt * 3, d_connector_num, total_rings, d_rings);
            hipDeviceSynchronize();

            hipMemcpy(h_rings, d_rings, total_rings * 6 * sizeof(int), hipMemcpyDeviceToHost);


            /*
            std::cout << "h_ring start" << std::endl;
            for(int i = 0; i < total_rings; ++i) {
                for(int j = 0; j < 6; ++j) {
                    std::cout << h_rings[i * 6 + j] << " ";
                }
                std::cout << std::endl;
            } 
            */

            // Add NO2
            int* d_c6rings;
            hipMalloc((void**) &d_c6rings, total_rings * 6 * sizeof(int));
            hipMemcpy(d_c6rings, h_rings, total_rings * 6 * sizeof(int), hipMemcpyHostToDevice);

            int * d_tnt;
            hipMalloc((void**) &d_tnt, total_rings * 15 * sizeof(int));
            addNO2<<<num_blocks_per_grid, num_threads_per_block>>>(d_c6rings, total_rings, d_cn, c_n_size, d_no, n_o_size, d_tnt);
            hipDeviceSynchronize();

            int* h_tnt = (int*) malloc(total_rings * 15 * sizeof(int));
            hipMemcpy(h_tnt, d_tnt, (total_rings * 15 * sizeof(int)), hipMemcpyDeviceToHost);


            /*
            for(int i = 0; i < total_rings; ++i) {
                for(int j = 0; j < 15; ++j) {
                    std::cout << h_tnt[i * 15 + j] << " ";
                }
                std::cout << std::endl;
            }
            */

            // 48 Mappings
            /*
            int * d_tnt_48;
            hipMalloc((void**) &d_tnt_48, total_rings  * 15 * 48 * sizeof(int));

            add48Mappings<<<num_blocks_per_grid, num_threads_per_block>>>(d_tnt, total_rings, d_tnt_48);
            hipDeviceSynchronize();
            int* h_tnt_48;
            hipMemcpy(h_tnt_48, d_tnt_48, (total_rings  * 15 * 48 * sizeof(int)), hipMemcpyDeviceToHost);
            */

            
            /*
            int* temp = (int*) malloc(total_rings * 48 * 15 * sizeof(int));
            for(int i = 0; i < total_rings * 48; ++i) {
                for(int k = 0; k < 15; ++k) {
                    temp[15 * i + k] = 1;
                }
            }
            h_tnt_48 = temp;
            */

            final_result_size = total_rings * 48;
            final_results = (int*) malloc(final_result_size * 15 * sizeof(int));

            for (int i = 0; i < total_rings; i++) {

                int* c6_rings = (int*) malloc(6 * sizeof(int));
                int* n3_array = (int*) malloc(3 * sizeof(int));
                int* o6_array = (int*) malloc(6 * sizeof(int));

                for(int j = 0; j < 6; ++j) {
                    c6_rings[j] = h_tnt[i * 15 + j];
                }

                for(int j = 0; j < 3; ++j) {
                    n3_array[j] = h_tnt[i * 15 + j + 6];
                }
            
                for(int j = 0; j < 6; ++j) {
                    o6_array[j] = h_tnt[i * 15 + j + 9];    
                }
                
                for(int j = 0; j < 48; ++j) {
                    for (int k = 0; k < NUM_TNT_VERTICES; k++) {
                        if(0 <= k && k <= 5) //c6
                        {
                            final_results[k * final_result_size + (48 * i + j)] = c6_rings[k];
                        }
                        
                        else if(6 <= k && k <= 8) //n3
                        {
                            final_results[k * final_result_size + (48 * i + j)] = n3_array[k % 3];
                        }
                        
                        else // o6
                        {
                            //std::cout << "o6 index is " <<  k % 6 << std::endl;
                            //std::cout << "o6 arrays gives " <<  o6_array[k % 6] << std::endl;
                            final_results[k * final_result_size + (48 * i + j)] = o6_array[k % 6];
                        }
                    }
                }

                for(int j = 0; j < final_result_size; j +=8) {
                    for(int k = 0; k < 6; ++k) {
                        //h_tnt_48[k * final_result_size + (48 * i + j)] = h_tnt_48[];
                    }
                }
            }

            //final_results = h_tnt_48;
            // freememory
            hipFree(d_cn);
            hipFree(d_valid_cn_len);
            hipFree(d_no);
            hipFree(d_valid_no);
            hipFree(d_3_tuple);
            hipFree(d_valid_no);
            hipFree(d_c_c);
            hipFree(d_connector_num);
            hipFree(d_rings);
            hipFree(d_tnt);
            hipFree(d_c6rings);

            free(h_rings);
            free(h_valid_cn);
            free(h_3_tuple);
            free(h_connector_num);
            free(h_tnt);

            /*
            for (int i = 0; i < total_rings; i++) {
                    for(int j = 0; j < 48; ++j) {
                        for (int k = 0; k < NUM_TNT_VERTICES; k++) {

                            h_tnt_48[k * final_result_size + (48 * i + j)] = 0;
                    }
                }
            }

            for(int i = 0; i < total_rings; ++i) {

                int* c6_rings = (int*) malloc(6 * sizeof(int));
                int* n3_array = (int*) malloc(3 * sizeof(int));
                int* o6_array = (int*) malloc(6 * sizeof(int));

                for(int j = 0; j < 6; ++j) {
                    c6_rings[j] = h_tnt[i * 15 + j];
                }

                for(int j = 0; j < 3; ++j) {
                    n3_array[j] = h_tnt[i * 15 + j + 6];
                }
            
                for(int j = 0; j < 6; ++j) {
                    o6_array[j] = h_tnt[i * 15 + j + 9];    
                }

                for (int j = 0; j < 48; ++j) {
                    for (int k = 0; k < 15; ++k) {
                        h_tnt[k * final_result_size + j] = 0;
                    }
                }

            */
                /*

                for(int j = 0; j < 6; ++j) {
                    h_tnt_48[48 * i + ((j + shift) % 6) +  k] = c6_rings[j];
                }

                for(int shift = 0; shift < 6; ++shift) {
                    for(int j = 0; j < 6; ++j) {
                        for(int k = 0; k < 8; ++k) {
                            h_tnt_48[48 * i + ((j + shift) % 6) +  k] = c6_rings[j];
                        }
                    }
                }
                */

                /*

                for(int shift = 0; shift < 3; ++shift) {
                    for(int j = 0; j < 3; ++j) {
                        for(int k = 0; k < 8; ++k) {
                            h_tnt_48[(48 * 15 * i + 6) + ((j + shift) % 3) +  k * 15] = n3_array[j];
                        }
                    }
                }
            
                for(int j = 0; j < 3; ++j) {
                    for(int k = 0; k < 2; ++k) {
                        if(j == 0) {
                            for(int row = 0; row < 2; ++row) {
                                for(int pat = 0; pat < 4; ++pat) {
                                    h_tnt_48[(48 * 15 * i + 9 + j * 2) + (row) + pat * 15] = o6_array[2 * j + k];
                                }
                            }
                        } else if(j == 1) {
                            for(int row = 0; row < 4; ++row) {
                                for(int pat = 0; pat < 2; ++pat) {
                                    h_tnt_48[(48 * 15 * i + 9 + j * 2) + (row % 2) +  (row * 2 + pat) * 15] = o6_array[2 * j + k];
                                }
                            }
                        } else {
                            for(int row = 0; row < 8; ++row) {
                                h_tnt_48[(48 * 15 * i + 9 + j * 2) +  row * 15 + row % 2] = o6_array[2 * j + k];
                            }
                        }
                    }
                }
                
            }
            */


            /*
            int *h_valid_no = (int*) malloc(n_o_size * sizeof(int)), *h_valid_no_len = (int*) malloc(sizeof(int));
            hipMemcpy(h_valid_no, d_valid_no, n_o_size * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(h_valid_no_len, d_valid_no_len, sizeof(int), hipMemcpyDeviceToHost);

            //std::cout << "valid_no2_len is " << h_valid_no_len[0] << std::endl;

            for(int i = 0; i < h_valid_no_len[0]; ++i)
                std::cout << h_valid_no[i] << " ";
            std::cout << std::endl;

            // Assume that all N are valid
            h_valid_no_len[0] = 3;
            h_valid_no[0] = 6; h_valid_no[1] = 7; h_valid_no[2] = 8;
            
            int *d_valid_cn, *d_valid_cn_len;
            //hipMalloc((void**) &d_cn, c_n_size * sizeof(int));
            hipMalloc((void**) &d_valid_cn, c_n_size * sizeof(int));
            hipMalloc((void**) &d_valid_cn_len, sizeof(int));
            hipMemcpy(d_cn, c_n, n_o_size * sizeof(int), hipMemcpyHostToDevice);

            int* h_valid_c = (int*) malloc(3 * sizeof(int));
            h_valid_c[0] = 0; h_valid_c[1] = 2; h_valid_c[2] = 4;

            int* d_valid_c;
            hipMalloc((void **) &d_valid_c, 3 * sizeof(int));
            hipMemcpy(d_valid_c, h_valid_c, 3 * sizeof(int), hipMemcpyHostToDevice);

            // for each 3-tuple, return the number of possible connectors between ab, bc, and ac
            // ex) output: c1c2c3 1 1 2 means that there are total possible c1-X-c2-Y-c3-Z1 and c1-X-c2-Y-c3-Z2

            int* d_valid_c_ring;
            hipMalloc((void **) &d_valid_c_ring, 3 * sizeof(int));

            int *d_c_c;
            hipMalloc((void**) &d_c_c, c_c_size * sizeof(int)); 
            hipMemcpy(d_c_c, c_c, c_c_size * sizeof(int), hipMemcpyHostToDevice);
            
            int *d_c_c_connectors;
            hipMalloc((void**) &d_c_c_connectors, c_c_size * sizeof(int)); 

            int *d_ring_num;
            hipMalloc((void**) &d_ring_num, c_c_size * sizeof(int));

            int *h_c_c_connectors = (int*) malloc(c_c_size * sizeof(int));
            int *h_ring_num = (int*) malloc( sizeof(int));
            int *h_valid_c_ring = (int*) malloc(3 * sizeof(int));

            // int* d_cc, int cc_len, int* d_cn, int cn_len, int* d_out_c_part, int* d_out_connector, int* tuple_num
            // tuple_connector<<<num_blocks_per_grid, num_threads_per_block>>>(d_c_c, c_c_size, d_valid_c, 3, h_valid_c_ring, d_c_c_connectors, d_ring_num);
            // hipDeviceSynchronize();

            hipMemcpy(h_c_c_connectors, d_c_c_connectors, c_c_size * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(h_valid_c_ring, d_valid_c_ring, 3 * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(h_ring_num, d_ring_num, sizeof(int), hipMemcpyDeviceToHost);

            std::cout << "h_valid_c_ring is " << h_ring_num[0] << std::endl;
            
            for(int i = 0; i < h_ring_num[0]; ++i) {
                for(int j = 0; j < 3; ++j) {
                    std::cout << h_valid_c_ring[i * 3 + j] << " "  << h_c_c_connectors[i * 3 + j] << std::endl;
                }
                std::cout << std::endl;
            }
            std::cout << std::endl;
            

            // create discrete rings of 6 carbon atoms

            int *d_connector_num;
            hipMalloc((void**) &d_connector_num, 3 * sizeof(int));

            int d_rings_len = 1;
            int *h_connector_num = (int*) malloc(3 * sizeof(int));

            //int* d_cc, int cc_len, int* d_c_tuple, int tuple_len, int total_ring_num, int* d_out_ring
            createRing_fin<<<num_blocks_per_grid, num_threads_per_block>>>(d_c_c, c_c_size, d_valid_c, 3, d_connector_num);
            hipDeviceSynchronize();
            hipMemcpy(h_connector_num, d_connector_num, 3 * sizeof(int), hipMemcpyDeviceToHost);

            std::cout << "number of each connectors called createRing_fin" << std::endl;
            int total_rings = 1;
            for(int i = 0; i < 3; i += 3) {
                for(int j = 0; j < 3; ++j) {
                    total_rings *= h_connector_num[i * 3 + j];
                }
            }

            for(int i = 0; i < 3; ++i) {
                std::cout << h_connector_num[i] << " ";
            }
            std::cout << std::endl;

            hipMemcpy(d_connector_num, h_connector_num, 3 * sizeof(int), hipMemcpyHostToDevice);

            int *d_rings;
            hipMalloc((void**) &d_rings, total_rings * 6 * sizeof(int));

            int *h_rings = (int*) malloc(total_rings * 6 * sizeof(int));

            createRing<<<num_blocks_per_grid, num_threads_per_block>>>(d_c_c, c_c_size, d_valid_c, d_rings_len, d_connector_num, 3, total_rings, d_rings);
            hipDeviceSynchronize();

            hipMemcpy(h_rings, d_rings, total_rings * 6 * sizeof(int), hipMemcpyDeviceToHost);

            std::cout << "h_ring started" << std::endl;
            for(int i = 0; i < total_rings; ++i) {
                for(int j = 0; j < 6; ++j) {
                    std::cout << h_rings[i * 6 + j] << " ";
                }
                std::cout << std::endl;
            } 


            */


            // add the corresponding N

            // add the corresponding O

            // permute to create 48 rings

            // store it to final_results and final_result_size

            // free all memory

            /*
            // maksing unique c starting points
            int * d_c_c; 
            hipMalloc((void **) &d_c_c, c_c_size * sizeof(int)); 
            
            bool *h_unique_c_mask, *h_c_visited;
            h_unique_c_mask = (bool*) malloc((c_c_size / 2) * sizeof(bool));
            h_c_visited = (bool*) malloc((c_c_size / 2) * sizeof(bool));
            
            for(int i = 0; i < c_c_size / 2; ++i) {
                h_c_visited[i] = false;
            }

            bool* d_unique_c_mask, *d_c_visited; 
            hipMalloc((void **) &d_unique_c_mask, c_c_size / 2 * sizeof(bool));
            hipMalloc((void **) &d_c_visited, c_c_size / 2 * sizeof(bool));

            hipMemcpy(d_c_c, c_c, c_c_size * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_unique_c_mask, h_unique_c_mask, (c_c_size / 2 ) * sizeof(bool), hipMemcpyHostToDevice);
            hipMemcpy(d_c_visited, h_c_visited, (c_c_size / 2 ) * sizeof(bool), hipMemcpyHostToDevice);

            unique_c_mask<<<num_blocks_per_grid, num_threads_per_block>>>(d_c_c, c_c_size / 2, d_c_visited, d_unique_c_mask); 
            hipDeviceSynchronize();
            hipMemcpy(h_unique_c_mask, d_unique_c_mask, (c_c_size / 2) * sizeof(bool), hipMemcpyDeviceToHost);

            for(int i = 0; i < c_c_size / 2; ++i) {
                std::cout << h_unique_c_mask[i] << std::endl; 
            }
            
            hipFree(d_c_c);
            hipFree(d_unique_c_mask);
            hipFree(d_c_visited);
            free(h_unique_c_mask);
            free(h_c_visited);
            */
}

/*
Input : cn array and no array
Output: id of n with 2 oxygens
*/
__global__ void valid_n(int* d_no, int no_len, int* d_valid_no, int* d_valid_no_len) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    // bring in c_n and n_o array
    int valid_no_len = 0;
    int cnt = 0;
    for(int i = tid; i < no_len; i += num_threads) {
        cnt = 0; 
        for(int j = 0; j < no_len; ++j) {
            if(d_no[i] == d_no[j]) {
                ++cnt;
            }
        }
        
        if(cnt == 2) {
            //printf("%d has 2 counts\n", d_no[i]);
            d_valid_no[i] = d_no[i];
            ++valid_no_len; 
        }
    }
    
    d_valid_no_len[0] = valid_no_len;
}

__global__ void unique_c_mask(int *d_c_c, int arr_len, bool* d_visited, bool *d_result_mask) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int num_threads = blockDim.x + gridDim.x;

    for(int i = tid; i < arr_len; i += num_threads) {
        for(int j = 0; j < arr_len; ++j) {
            d_result_mask[j] = true;
            if(d_c_c[i] == d_c_c[j]) {
                d_result_mask[j] = false;
            }

            /* 
            if(!d_visited[j]) {
                d_visited[j] = true;
                if(d_c_c[i] != d_c_c[j]) {
                    printf("Diff %d vs %d\n", d_c_c[i], d_c_c[j]);
                    d_result_mask[j] = true;
                } else {
                    d_result_mask[j] = false;
                }
            } 
            */
        }
    }
}

//printf("a is %d\n", a); printf("b is %d\n", b); printf("c is %d\n", c);

/*
__global__ void tuple_connector(int* d_cc, int cc_len, int* d_cn, int cn_len, int* d_out_c_part, int* d_out_connector, int* tuple_num) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;
    int cc_len_half = cc_len / 2;

    int valid_ring = 0;

    for(int i = tid; i < cn_len; i += num_threads) {
        for(int j = i + 1; j < cn_len; ++j) {
            for(int k = j + 1; k < cn_len; ++k) {
                int a = d_cn[i], b = d_cn[j], c = d_cn[k];
                int a_b_cnt = 0, b_c_cnt = 0, c_a_cnt = 0; 
                
                for(int l = 0; l < cc_len; ++l) {
                    if(d_cc[l] == a || d_cc[l] == b || d_cc[l] == c) {
                        //printf("First continue %d\n", d_cc[l]);
                        continue;
                    }

                    for(int m = l + 1; m < cc_len; ++m) {
                        //printf("Second continue\n");
                        if(d_cc[m] == a || d_cc[m] == b || d_cc[m] == c) {
                            continue;
                        }

                        //printf("%d vs %d\n", d_cc[l], d_cc[m]);
                        if(d_cc[l] == d_cc[m]) {
                            if(d_cc[(l + cc_len_half) % cc_len] == a && d_cc[(m + cc_len_half) % cc_len] == b || d_cc[(l + cc_len_half) % cc_len] == b && d_cc[(m + cc_len_half) % cc_len] == a) {
                                printf("a_b_cnt is going up by %d and %d\n", l, m);
                                ++a_b_cnt;
                            }

                            if(d_cc[(l + cc_len_half) % cc_len] == b && d_cc[(m + cc_len_half) % cc_len] == c || d_cc[(l + cc_len_half) % cc_len] == c && d_cc[(m + cc_len_half) % cc_len] == b) {
                                printf("b_c_cnt is going up by %d and %d\n", l, m);
                                ++b_c_cnt;
                            }
                        
                            if(d_cc[(l + cc_len_half) % cc_len] == c && d_cc[(m + cc_len_half) % cc_len] == a || d_cc[(l + cc_len_half) % cc_len] == a && d_cc[(m + cc_len_half) % cc_len] == c) {
                                printf("c_a_cnt is going up by %d and %d\n", d_cc[l], d_cc[m]);
                                ++c_a_cnt;
                            }
                        }
                        //printf("a_b is %d b_c is %d c_a is %d\n", a_b_cnt, b_c_cnt, c_a_cnt);
                    }
                }
                int total = a_b_cnt * b_c_cnt * c_a_cnt;
                printf("total value is %d\n", total);
                if(total == 0) {
                    printf("why?\n");
                    continue;
                } else {
                    d_out_c_part[valid_ring * 3 + 0] = a;
                    d_out_c_part[valid_ring * 3 + 1] = b;
                    d_out_c_part[valid_ring * 3 + 2] = c;
                    d_out_connector[valid_ring * 3 + 0] = a_b_cnt;
                    d_out_connector[valid_ring * 3 + 1] = b_c_cnt;
                    d_out_connector[valid_ring * 3 + 2] = c_a_cnt;
                    ++valid_ring;
                    tuple_num[0] = total;

                    //for(int tup1 = 0; tup1 < a_b_cnt; ++tup1) {
                    //    for(int tup2 = 0; tup2 < b_c_cnt; ++tup2) {
                    //        for(int tup3 = 0; tup3 < c_a_cnt; ++tup3) {
                    //            d_out_c_part[valid_ring * 3 + 1] = a;
                    //            d_out_c_part[valid_ring * 3 + 3] = b;
                    //            d_out_c_part[valid_ring * 3 + 5] = c;
                    //            d_out_connector[a_b_cnt * 3 + 0] = a_b_cnt;
                    //            d_out_connector[b_c_cnt * 3 + 2] = b_c_cnt;
                    //            d_out_connector[c_a_cnt * 3 + 4] = c_a_cnt;
                    //            ++valid_ring;
                    //        }
                    //    }
                    //}
    
                }
            } 
        }
    }
    //tuple_num[0] = valid_ring;
}
*/

/*
for(int shift = 0; shift < 3; ++shift) {
    for(int j = 0; j < 6; ++j) {
        for(int k = 0; k < 8; ++k) {
            h_tnt_48[(48 * 15 * i + 9) + j +  k * 15] = n3_array[j];
        }
    }
}

                
for(int j = 0; j < 2; ++j) {
    for(int k = 0; k < 2; ++k) {
        for(int l = 0; l < 2; ++l) {
            for(int eachrow = 0; eachrow < 8; ++eachrow) {
                h_tnt_48[(48 * 15 * i + 9) + ((j + shift) % 3) +  eachrow * 15] = o6_array[4 * j + 2 * k + l];
            }
        }
    }
}
*/
