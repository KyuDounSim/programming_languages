/**
 *	Demo code of Cuda programming lecture
 *	
 *	This programme illustrates how warp divergence may influence the performance of CUDA programme
 *
 *
 */

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <sys/time.h>

#define HALF_BLOCK_SIZE 512
#define BLOCK_SIZE 1024
#define LOOP_NUM 1024

//Kernel1 (has warp divergence)
__global__ void kernel1(int *A, int *B)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;	

	if (i % 2 == 0)
	{
		/*Execution Path 1: thread 0, 2, 4, 6...30... reach here*/
		
		int lower_bound = B[i];
		int tmp = 0;
		
		//do some computation to make this execution path long enough
		for (int j = lower_bound; j < lower_bound+LOOP_NUM; j += 2)	
			tmp += j;
		A[i] += tmp;
	}
	else
	{
		/*Execution Path 2: thread 1, 3, 5, 7...31... reach here*/

		int lower_bound = B[i];
		int tmp = 0;
		
		//do some computation to make this execution path long enough
		for (int j = lower_bound; j < lower_bound+LOOP_NUM; j += 2)	
			tmp += j;

		A[i] -= tmp;
	}
		
 	/*even threads and odd threads go back to the same exexution path*/
}

//Kerne2 (does not have warp divergence)
__global__ void kernel2(int *A, int *B)
{
	int base = blockIdx.x*blockDim.x;	
	
	if (threadIdx.x < HALF_BLOCK_SIZE)
	{
		/*Execution Path 1: the first half threads of a block reach here*/

		int even_index = base + threadIdx.x*2;
		
		int lower_bound = B[even_index];
		int tmp = 0;
		
		//Do some computation
		for (int j = lower_bound; j < lower_bound+LOOP_NUM; j += 2)	
			tmp += j;

		A[even_index] += tmp;
	}
	else
	{
		/*Execution Path 2: the second half threads of a block reach here*/

		int odd_index = base + (threadIdx.x - HALF_BLOCK_SIZE)*2 +1;
		
		int lower_bound = B[odd_index];
		int tmp = 0;
		
		//Do some computation
		for (int j = lower_bound; j < lower_bound+LOOP_NUM; j += 2)	
			tmp += j;

		A[odd_index] -= tmp;
	}
}

int main()
{
	//Device and host memory pointers
	int *h_A, *h_B, *d_A, *d_B;
	
	int N = 33554432;
	int data_size = N*(sizeof(int));
	
	//Kernel configuration parameter
	int threads_per_block = BLOCK_SIZE;
	int blocks_per_grid = N / threads_per_block;
	
	//Time measurement
	timeval k1_start, k1_end, k2_start, k2_end;
	float k1_elapsed_time, k2_elapsed_time;
	
	//Allocate Host Memory
	h_A = (int*)malloc(data_size);
	h_B = (int*)malloc(data_size);
	
	//Allocate Device Memory
	hipMalloc((void**)&d_A, data_size);
	hipMalloc((void**)&d_B, data_size);
	
	//Initialization
	for (int i = 0; i < N; i++)
	{
		h_A[i] = i;
		h_B[i] = i;
	}
	
	//Memory copy from host to device
	hipMemcpy(d_A, h_A, data_size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, data_size, hipMemcpyHostToDevice);
	
	
	gettimeofday(&k1_start, NULL);
	
	//Invoke kernel1(has warp divergence)
	kernel1<<<blocks_per_grid, threads_per_block>>>(d_A, d_B);

	hipDeviceSynchronize();

	gettimeofday(&k1_end, NULL);


	gettimeofday(&k2_start, NULL);
	
	//Invoke kernel2(does not have warp divergence)
	kernel2<<<blocks_per_grid, threads_per_block>>>(d_A, d_B);

	hipDeviceSynchronize();

	gettimeofday(&k2_end, NULL);

	//Copy result back from device to host
	hipMemcpy(h_A, d_A, data_size, hipMemcpyDeviceToHost);

	k1_elapsed_time = 1000*(k1_end.tv_sec - k1_start.tv_sec) + (float)(k1_end.tv_usec - k1_start.tv_usec)/1000;
	k2_elapsed_time = 1000*(k2_end.tv_sec - k2_start.tv_sec) + (float)(k2_end.tv_usec - k2_start.tv_usec)/1000;
	
	printf("elapsed time of kernel function which has warp divergence: %.2f ms\n", k1_elapsed_time);
	printf("elapsed time of kernel function which has no warp divergence: %.2f ms\n", k2_elapsed_time);
	
	//Free device memory
	hipFree(d_A);
	hipFree(d_B);
	
	//Free host memory
	free(h_A);
	free(h_B);

	return 0;
}

