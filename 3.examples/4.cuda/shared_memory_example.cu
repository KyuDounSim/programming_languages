/**
 *	Demo code of Cuda programming lecture
 *	
 *	This programme illustrates the benefit of using shared memory
 *
 *
 */


#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <sys/time.h>

//Kernel function that does not use shared memory, each memory read is from global memory
 __global__ void compute_no_shared_memory(int *data)
{
     int tid = threadIdx.x;
     int* base = data + blockIdx.x * blockDim.x;
     int tmp = 0;
	
     //Do some computation 	
     for (int i = 0; i < tid; i++)
	     tmp += base[i];
     
     //Make sure all threads in a block have completed computation	
     //__syncthreads();	

     base[tid] = tmp + base[tid];
}

//Kernel function that utilizes shared memory
 __global__ void compute_use_shared_memory(int *data)
{
     int tid = threadIdx.x;
     int* base = data + blockIdx.x * blockDim.x;
     int tmp = 0;

     __shared__ int myblock[1024];

     // load data from global memory to shared memory
     myblock[tid] = base[tid];

     // ensure that all threads have loaded their values into
     // shared memory; Otherwise, one thread might be computing
     // on unitialized data.
     __syncthreads();

     //Do some computation 	
     for (int i = 0; i < tid; i++) 
	     tmp += myblock[i];   

     // write the result back to global memory
     base[tid] = tmp + myblock[tid];
}

int main()
{	
	//Host and device pointers
	int * h_data, *d_data;
	int N = 33554432;
	int data_size = N * sizeof(int);
	
	//Kernel configuration parameters
	int threads_per_block = 1024;
	int blocks_per_grid = N / threads_per_block;
	
	//For time measurement
	timeval start, end;
	float elapsed_time_use_shared_m;
	float elapsed_time_no_shared_m;
	
	//Host memory allocation
	h_data = (int*)malloc(data_size);
	
	//Device memory allocation
	hipMalloc((void**)&d_data, data_size);
	
	//Initialization
	for (int i = 0; i < N; i++)
		h_data[i] = i;
	
	//Memory copy from the host to the device
	hipMemcpy(d_data, h_data, data_size, hipMemcpyHostToDevice);
	
	//Start timer
	gettimeofday(&start, NULL);
	
	//Invoke the kernel that utilize shared memory
	compute_use_shared_memory<<<blocks_per_grid, threads_per_block>>>(d_data);
	
	//Wait for kernel execution
	hipDeviceSynchronize();
	
	//End timer
	gettimeofday(&end, NULL);
	
	//Calculate elapsed time
	elapsed_time_use_shared_m = 1000*(end.tv_sec-start.tv_sec) + (float)(end.tv_usec - start.tv_usec)/1000;
	
	//Copy data to device memory
	hipMemcpy(d_data, h_data, data_size, hipMemcpyHostToDevice);

	//Start timer
	gettimeofday(&start, NULL);
	
	//Invoke the kernel that does not use shared memory
	compute_no_shared_memory<<<blocks_per_grid, threads_per_block>>>(d_data);
	
	//Wait for kernel execution
	hipDeviceSynchronize();
	
	//End timer
	gettimeofday(&end, NULL);
	
	//Calculate time
	elapsed_time_no_shared_m = 1000*(end.tv_sec-start.tv_sec) + (float)(end.tv_usec - start.tv_usec)/1000;
	
	printf("elapsed time of kernel funtion that uses shared memory: %.2f ms\n", elapsed_time_use_shared_m);
	printf("elapsed time of kernel funtion that does not use shared memory: %.2f ms\n", elapsed_time_no_shared_m);
	
	//Free device and host memory
	free(h_data);
	hipFree(d_data);

	return 0;
}

